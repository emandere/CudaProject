
#include <hip/hip_runtime.h>
#include<ctime>
#include<iostream>
#include <cstdlib>
using namespace std;

#define BLOCK_SIZE 32

__global__ void gpuSum(int *prices,int *sumpricesout,int days,int seconds,int N)
{
    int currentday = blockIdx.x*blockDim.x + threadIdx.x;
    if(currentday<days)
    {
       int start = currentday * seconds;
       int end = (currentday * seconds)+seconds;
       sumpricesout[currentday]=0;
       for(int j=start;j<end;j++)
         sumpricesout[currentday]+=prices[j]; 
    }
}
int main()
{
   int days = 12;
   int seconds = 1000;
   int N = days*seconds;
   
   int * prices = new int[days*seconds];
   int  * sumpricesout = new int[days];
   int sumprices;
   for(int i=0;i<N;i++)
   {
      prices[i]=rand()%100;  
   }
   
  
   for(int i=0;i<N;i=i+seconds)
   {
      sumprices=0;
      for(int j=i;j<i+seconds;j++)
         sumprices+=prices[j];
      cout<<sumprices<<endl;
   }
   
   return 0;
}
