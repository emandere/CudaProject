#include "hip/hip_runtime.h"
#include<ctime>
#include<iostream>
#include <cstdlib>
using namespace std;

#define BLOCK_SIZE 64

__global__ void gpuSum(int *prices,int *sumpricesout,int days,int seconds,int N)
{
    int currentday = blockIdx.x*blockDim.x + threadIdx.x;
    if(currentday<days)
    {
       int start = currentday * seconds;
       int end = start+seconds;
       sumpricesout[currentday]=0;
       for(int j=start;j<end;++j)
         sumpricesout[currentday]+=prices[j]; 
    }
}
int main()
{
   int days = 12;
   int seconds = 1000;
   
   clock_t start;
   double duration; 
   start = std::clock();
    
   int N = days*seconds;
   
   int * prices = new int[days*seconds];
   int  * sumpricesout = new int[days];
   int  * sumpricesoutCPU = new int[days]; 
   int sumprices;
   for(int i=0;i<N;i++)
   {
      prices[i]=rand()%100;  
   }
   
  
   for(int i=0;i<N;i=i+seconds)
   {
      sumprices=0;
      for(int j=i;j<i+seconds;j++)
         sumpricesoutCPU[i]+=prices[j];
      
   }
    duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	cout <<i<< " " << duration <<"s"<< '\n'; 
    
   for(int i=0;i<days;i++)
   {
       cout<<sumpricesoutCPU[i]<<endl;
   }
    
   long sizePrices = N * sizeof(int);
   long sizeSumPrices = days * sizeof(int); 
   int *dPrices,*dSumPrices;
    
   hipMalloc(&dPrices,sizePrices);
   hipMalloc(&dSumPrices,sizeSumPrices);
    
   hipMemcpy(dPrices,prices,sizePrices,hipMemcpyHostToDevice); 
   hipMemcpy(dSumPrices,sumpricesout,sizeSumPrices,hipMemcpyHostToDevice); 
    
   gpuSum<<< 1,BLOCK_SIZE>>>(dPrices,dSumPrices,days,seconds,N);
   
   hipMemcpy(sumpricesout,dSumPrices,sizeSumPrices,hipMemcpyDeviceToHost); 
   cout<<"CUDA!"<<endl;
   for(int i=0;i<days;i++)
   {
       cout<<sumpricesout[i]<<endl;
   }
   
   hipFree(dPrices);
   hipFree(dSumPrices); 
   return 0;
}
