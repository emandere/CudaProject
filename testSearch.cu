
#include <hip/hip_runtime.h>
#include<ctime>
#include<iostream>
#include <cstdlib>
using namespace std;

#define BLOCK_SIZE 32

__global__ void gpuSum(int *prices,int *sumpricesout,int days,int seconds,int N)
{
    int currentday = blockIdx.x*blockDim.x + threadIdx.x;
    //if(currentday<days)
    //{
       int start = currentday * seconds;
       int end = start+seconds;
       sumpricesout[currentday]=end;
       for(int j=start;j<end;++j)
         sumpricesout[currentday]+=prices[j]; 
    //}
}
int main()
{
   int days = 32;
   int seconds = 1000;
   int N = days*seconds;
   
   int * prices = new int[days*seconds];
   int  * sumpricesout = new int[days];
   int sumprices;
   for(int i=0;i<N;i++)
   {
      prices[i]=rand()%100;  
   }
   
  
   for(int i=0;i<N;i=i+seconds)
   {
      sumprices=0;
      for(int j=i;j<i+seconds;j++)
         sumprices+=prices[j];
      cout<<sumprices<<endl;
   }
    
   long sizePrices = N * sizeof(int);
   long sizeSumPrices = days * sizeof(int); 
   int *dPrices,*dSumPrices;
    
   hipMalloc(&dPrices,sizePrices);
   hipMalloc(&dSumPrices,sizeSumPrices);
    
   hipMemcpy(&dPrices,prices,N,hipMemcpyHostToDevice); 
   hipMemcpy(&dSumPrices,sumpricesout,days,hipMemcpyHostToDevice); 
    
   gpuSum<<< 1,BLOCK_SIZE>>>(dPrices,dSumPrices,days,seconds,N);
   
   hipMemcpy(sumpricesout,dSumPrices,days,hipMemcpyDeviceToHost); 
   cout<<"CUDA!"<<endl;
   for(int i=0;i<days;i++)
   {
       cout<<sumpricesout[i]<<endl;
   }
   
   hipFree(dPrices);
   hipFree(dSumPrices); 
   return 0;
}
